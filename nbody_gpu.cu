// N-BODY SIMULATION Using GPUs
// Exact pair interactions with Leapfrog Integration
// Written by Ashwin Nayak, asnayak[at]ucsd.edu
// ----------------------------------------------------------
// Graphics Card : NVIDIA GTX 960M
// ----------------------------------------------------------
// References : 
// GPU GEMS 3 Documentation
// CUDA Sample Programs and Documentation
// ----------------------------------------------------------
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <fstream>
#include <string>
// CUDA Libraries
#include "hip/hip_runtime.h"

//#include "math_functions.h"
// Timer Libraries
#include <chrono>
//#include <ctime>

// GLOBAL PARAMETERS
#define PI 4*atan(1.)
#define G 4.498279e-3

// GLOBAL SIMULATION PARAMETERS
#define NPAR 10000

// GPU PARAMETERS
#define BLOCK_SIZE 512

struct Particle {
	float m, x, y, z, v_x, v_y, v_z;
};
// Input Data from File
void input_data(Particle *p) {
	std::ifstream input("plummer_init.dat");
	for (int i = 0; i < NPAR; i++) {
		input >> p[i].x
			>> p[i].y
			>> p[i].z
			>> p[i].v_x
			>> p[i].v_y
			>> p[i].v_z;
		p[i].m = 1.0e11/NPAR;
	}
	input.close();
}

// Leapfrog Integration in Device
__global__ void leapfrog(Particle *p, float dt) {
	// Each particle decided by thread index 
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	
	if (i<NPAR) {
		//// ---------------- Position Verlet algorithm -------------------
		//// Update position 1/2
		p[i].x += 0.5 * dt * p[i].v_x;
		p[i].y += 0.5 * dt * p[i].v_y;
		p[i].z += 0.5 * dt * p[i].v_z;

		// Compute Acceleration
		float a_x = 0.; float a_y = 0.; float a_z = 0.;
			for (int j = 0; j < NPAR; j++) {
				if (i != j) {
					float r_x = p[i].x - p[j].x;
					float r_y = p[i].y - p[j].y;
					float r_z = p[i].z - p[j].z;
					float r_2 = r_x*r_x + r_y*r_y + r_z*r_z;
					float inv_r = rsqrtf(r_2); 
					a_x -=  p[j].m * r_x * inv_r * inv_r * inv_r;
					a_y -=  p[j].m * r_y * inv_r * inv_r * inv_r;
					a_z -=  p[j].m * r_z * inv_r * inv_r * inv_r;
				}
			}
		// Update Velocity
			p[i].v_x += dt * G * a_x;
			p[i].v_y += dt * G * a_y;
			p[i].v_z += dt * G * a_z;

		// Update Position 2/2
			p[i].x += 0.5 * dt * p[i].v_x;
			p[i].y += 0.5 * dt * p[i].v_y;
			p[i].z += 0.5 * dt * p[i].v_z;
		// --------------------------------------------------------------
	 } 
}

void output_data(Particle *p, int n) {
	std::string file_name = "pl_";

	file_name += std::to_string(n) + ".dat";
	std::ofstream output(file_name.c_str());
	//output  << p[i].m << "\t"
	for (int i = 0; i < NPAR; i++) {
		output << p[i].x << "\t"
			<< p[i].y << "\t"
			<< p[i].z << "\t"
			<< p[i].v_x << "\t"
			<< p[i].v_y << "\t"
			<< p[i].v_z << "\n";

	}
}

void PressEnterToContinue() {
	std::cout << "Press ENTER to continue... ";
	std::cin.ignore(std::numeric_limits <std::streamsize> ::max(), '\n');
}

// MAIN PROGRAM
int main() {	
	
	int nt, dsnap;
	float t, dt;
	hipError_t cudaStatus;

	// Allocate Arrays
	int num_bytes = NPAR * sizeof(Particle);
	Particle *p = new Particle[NPAR];

	// Read Input Parameters from file 
	input_data(p);
	
	// Allocate Memory on Device
	cudaStatus = hipSetDevice(0);
	Particle *dev_p = new Particle[NPAR];
	cudaStatus = hipMalloc(&dev_p, num_bytes);
	if (cudaStatus != hipSuccess) fprintf(stderr, "hipMalloc failed!");

	int NBlocks = (NPAR + BLOCK_SIZE - 1) / BLOCK_SIZE;
	// Simulation Parameters 
	t = 0.;		dt = 0.005;
	nt = 3000;	dsnap = 12;
	
	// Start Timer
	auto start = std::chrono::system_clock::now();

	// Time Loop
	for (int it = 0; it < nt; it++) {
		t = t + dt;

		// Display Progress
		if (it%100==0)
		std::cout << "Iteration:" << it+1 << "\t Time: " << t << std::endl;
		
		// Copy memory to device
		cudaStatus = hipMemcpy(dev_p, p, num_bytes, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "hipMemcpy failed!");
		
		// March all particles in time
		leapfrog<<<NBlocks,BLOCK_SIZE>>>(dev_p,dt);
		
		// Copy memory back to Host
		cudaStatus = hipMemcpy(p, dev_p, num_bytes,hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "cudaMemcpy2 failed!");
		
		// Output every dsnap iterations
		if ((it+1)%dsnap == 0) output_data(p, it+1);
		
	}
	
	// Stop Timer
	auto finish = std::chrono::system_clock::now();
	
	// Display Timer info
	std::cout << "Time elapsed : " 
		<< std::chrono::duration_cast<std::chrono::seconds>(finish-start).count()/60.0
		<< " minutes\n";
	std::cout << "Avg time per iteration : "
		<< (std::chrono::duration_cast<std::chrono::seconds>(finish - start).count())/(float)nt
		<< " seconds\n";

	hipFree(dev_p);
	delete[] p;

	PressEnterToContinue(); // Windows user's dilemma
	return 0;
}
